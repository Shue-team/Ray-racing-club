#include "hip/hip_runtime.h"
#include "Hittable/Sphere.h"
#include "Renderer.h"
#include "CommonMath.h"

inline __device__ void writeColor(uchar8* pixelPtr, const Color& color, int samplesPerPixel) {
    float scale = 1.0f / samplesPerPixel;

    pixelPtr[0] = (uchar8) (256 * clamp(color[0] * scale, 0.0f, 0.999f));
    pixelPtr[1] = (uchar8) (256 * clamp(color[1] * scale, 0.0f, 0.999f));
    pixelPtr[2] = (uchar8) (256 * clamp(color[2] * scale, 0.0f, 0.999f));
}

__global__ void sampleRender(uchar8* colorData, const Camera* cam,
                             const Hittable* world, int samplesPerPixel) {
    extern __shared__ Color pixelColor[];
    pixelColor[0][0] = pixelColor[0][1] = pixelColor[0][2] = 0;

    __syncthreads();

    unsigned int i = blockIdx.x;
    unsigned int j = blockIdx.y;

    float u = (float) i / (gridDim.x - 1);
    float v = (float) (j - gridDim.y + 1) / (gridDim.y - 1);
    Ray ray = cam->getRay(u, v);
    HitRecord record;

    Color currSample(0.5, 0.5, 0.5);
    /*if (world->hit(ray, 0, infinity, record)) {
        currSample = 0.5f * (record.normal + Color(1.0f, 1.0f, 1.0f));

    } else {
        Vector3D unitDir = ray.direction().normalized();
        float t = 0.5f * (unitDir.y() + 1.0f);
        currSample = (1.0f - t) * Color(1.0f, 1.0f, 1.0f) + t * Color(0.5f, 0.7f, 1.0f);
    }*/
    pixelColor[0].atomicAddVec(currSample);

    __syncthreads();

    if (threadIdx.x == 0) {
        uchar8* pixelPtr = colorData + 3 * (j * gridDim.x + i);
        writeColor(pixelPtr, pixelColor[0], samplesPerPixel);
    }
}

Renderer::Renderer(int imgWidth, int imgHeight, int samplesPerPixel) {
    mImgWidth = imgWidth;
    mImgHeight = imgHeight;
    mSamplesPerPixel = samplesPerPixel;

    size_t colorBuffSize = 3 * imgWidth * imgHeight;
    hipMalloc(&mColorBuff_d, colorBuffSize * sizeof(uchar8));
    mColorBuff_h = new uchar8[colorBuffSize * sizeof(uchar8)];

    mWorld = new Sphere(Vector3D(-0.5f, 0.0f, -1.0f), 0.5f);
}

uchar8* Renderer::render(const Camera* camera) {
    dim3 gridDim(mImgWidth, mImgHeight);
    sampleRender<<<gridDim, mSamplesPerPixel, sizeof(Color)>>>(mColorBuff_d, camera, mWorld, mSamplesPerPixel);

    size_t colorBuffSize = 3 * mImgWidth * mImgHeight;
    hipMemcpy(mColorBuff_h, mColorBuff_d, colorBuffSize * sizeof(char), hipMemcpyDeviceToHost);
    return mColorBuff_h;
}

Renderer::~Renderer() {
    delete mWorld;
    hipFree(mColorBuff_d);
    //hipFree(mCharBuff_h); //TODO: Uncomment when renderer instance will become dynamically allocated
}
