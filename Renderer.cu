#include "hip/hip_runtime.h"
#include "Hittable/Sphere.h"
#include "Renderer.h"
#include "Common/Math.h"

#include <iostream>

inline __device__ void writeColor(uchar8* pixelPtr, const Color& color, int samplesPerPixel) {
    float scale = 1.0f / samplesPerPixel;

    pixelPtr[0] = (uchar8) (256 * clamp(color[0] * scale, 0.0f, 0.999f));
    pixelPtr[1] = (uchar8) (256 * clamp(color[1] * scale, 0.0f, 0.999f));
    pixelPtr[2] = (uchar8) (256 * clamp(color[2] * scale, 0.0f, 0.999f));
}

__device__ Color getColor(const Ray& ray, Hittable* const* world) {
    HitRecord record;

    Color color;
    if ((*world)->hit(ray, 0, infinity, record)) {
        color = 0.5f * (record.normal + Color(1.0f, 1.0f, 1.0f));

    } else {
        Vector3D unitDir = ray.direction().normalized();
        float t = 0.5f * (unitDir.y() + 1.0f);
        color = (1.0f - t) * Color(1.0f, 1.0f, 1.0f) + t * Color(0.5f, 0.7f, 1.0f);

    }
    return color;
}

__global__ void pixelRender(int imgWidth, int imgHeight, int samplesPerPixel,
                             uchar8* colorData, hiprandState* randStateArr,
                             const Camera* cam, Hittable* const* world) {

    uint32 x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32 y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= imgWidth || y >= imgHeight) { return; }

    uint32 pixelIdx = imgWidth * y + x;
    hiprandState localRandState = randStateArr[pixelIdx];

    uint32 yMapped = imgHeight - y - 1;

    Color pixelColor;
    for (int i = 0; i < samplesPerPixel; i++) {
        float xDisturbed = x + 1 - hiprand_uniform(&localRandState);
        float yDisturbed = yMapped + 1 - hiprand_uniform(&localRandState);

        float u = xDisturbed / (float) (imgWidth - 1);
        float v = yDisturbed / (float) (imgHeight - 1);

        Ray ray = cam->getRay(u, v);
        pixelColor += getColor(ray, world);
    }

    writeColor(&colorData[3 * pixelIdx], pixelColor, samplesPerPixel);
}

__global__ void initRandomState(int imgWidth, int imgHeight, uint32 firstSeed,
                           hiprandState* randStateArr) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if((x >= imgWidth) || (y >= imgHeight)) return;

    uint32 pixelIdx = y * imgWidth + x;
    hiprand_init(firstSeed + pixelIdx, 0, 0, &randStateArr[pixelIdx]);
}

__global__ void createWorld(Hittable** world) {
    Hittable* sphere = new Sphere(Vector3D(0.0f, 0.0f, -1.0f), 0.5f);
    *world = sphere;
}

__global__ void destroyWorld(Hittable** world) {
    delete *world;
}

Renderer::Renderer(const RenderInfo& renderInfo) {
    mImgWidth = renderInfo.imgWidth;
    mImgHeight = renderInfo.imgHeight;
    mSamplesPerPixel = renderInfo.samplesPerPixel;

    mThreadBlockWidth = renderInfo.threadBlockWidth;
    mThreadBlockHeight = renderInfo.threadBlockHeight;

    mColorDataSize = 3 * mImgWidth * mImgHeight;
    hipMalloc(&mColorData_d, mColorDataSize * sizeof(uchar8));
    mColorData_h = new uchar8[mColorDataSize];

    hipMalloc(&mWorld_d, sizeof(Hittable*));
    createWorld<<<1, 1>>>(mWorld_d);

    hipMalloc(&mRandStateArr, mImgWidth * mImgHeight * sizeof(hiprandState));

    uint32 seed = (uint32)time(nullptr);

    int gridWidth = (mImgWidth + mThreadBlockWidth - 1) / mThreadBlockWidth;
    int gridHeight = (mImgHeight + mThreadBlockHeight - 1) / mThreadBlockHeight;

    dim3 gridDim(gridWidth, gridHeight);
    dim3 blockDim(mThreadBlockWidth, mThreadBlockHeight);

    initRandomState<<<gridDim, blockDim>>>(mImgWidth, mImgHeight, seed, mRandStateArr);
}

uchar8* Renderer::renderRaw(const Camera* camera) {
    clock_t start, stop;
    start = clock();

    int gridWidth = (mImgWidth + mThreadBlockWidth - 1) / mThreadBlockWidth;
    int gridHeight = (mImgHeight + mThreadBlockHeight - 1) / mThreadBlockHeight;

    dim3 gridDim(gridWidth, gridHeight);
    dim3 blockDim(mThreadBlockWidth, mThreadBlockHeight);

    pixelRender<<<gridDim, blockDim>>>(mImgWidth, mImgHeight, mSamplesPerPixel,
                                        mColorData_d, mRandStateArr,
                                        camera, mWorld_d);

    hipMemcpy(mColorData_h, mColorData_d, mColorDataSize * sizeof(uchar8), hipMemcpyDeviceToHost);

    stop = clock();
    double timerSeconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cout << "took " << timerSeconds << " seconds.\n";

    return mColorData_h;
}

Renderer::~Renderer() {
    destroyWorld<<<1, 1>>>(mWorld_d);

    hipFree(mWorld_d);
    hipFree(mColorData_d);
    hipFree(mRandStateArr);

    delete mColorData_h;
}